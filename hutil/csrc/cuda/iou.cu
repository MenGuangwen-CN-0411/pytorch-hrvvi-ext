#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>

#include "cuda_helpers.h"

#define GET_BLOCKS(block_size, n)                                              \
    ((static_cast<int>(n) + block_size - 1) / block_size)

template <typename T> __device__ inline float iou_11(const T *a, const T *b) {
    T left = max(a[0], b[0]), right = min(a[2], b[2]);
    T top = max(a[1], b[1]), bottom = min(a[3], b[3]);
    T width = max(right - left, (T)0), height = max(bottom - top, (T)0);
    T interS = width * height;
    T Sa = (a[2] - a[0]) * (a[3] - a[1]);
    T Sb = (b[2] - b[0]) * (b[3] - b[1]);
    return interS / (Sa + Sb - interS);
}

template <typename T>
__device__ inline void iou_11_backward(T *dbox1, T *dbox2, const T dout,
                                       const T *box1, const T *box2,
                                       const T out) {
    if (out == 0) {
        return;
    }

    T ix1 = box1[0];
    T iy1 = box1[1];
    T ix2 = box1[2];
    T iy2 = box1[3];
    T iw = ix2 - ix1;
    T ih = iy2 - iy1;
    T iarea = iw * ih;

    T jx1 = box2[0];
    T jy1 = box2[1];
    T jx2 = box2[2];
    T jy2 = box2[3];
    T jw = jx2 - jx1;
    T jh = jy2 - jy1;
    T jarea = jw * jh;

    T xx1 = std::max(ix1, jx1);
    T yy1 = std::max(iy1, jy1);
    T xx2 = std::min(ix2, jx2);
    T yy2 = std::min(iy2, jy2);
    T w = std::max(static_cast<T>(0.0), xx2 - xx1);
    T h = std::max(static_cast<T>(0.0), yy2 - yy1);
    T inter_area = w * h;
    T union_area = iarea + jarea - inter_area;

    T darea = dout * inter_area / (union_area * union_area);

    atomicAdd(dbox1, ih * darea);
    atomicAdd(dbox1 + 1, iw * darea);
    atomicAdd(dbox1 + 2, -ih * darea);
    atomicAdd(dbox1 + 3, -iw * darea);

    atomicAdd(dbox2, jh * darea);
    atomicAdd(dbox2 + 1, jw * darea);
    atomicAdd(dbox2 + 2, -jh * darea);
    atomicAdd(dbox2 + 3, -jw * darea);

    T dinter = dout * (inter_area + union_area) / (union_area * union_area);
    T dw = h * dinter;
    T dh = w * dinter;

    if (ix1 >= jx1) {
        atomicAdd(dbox1, -dw);
    } else {
        atomicAdd(dbox2, -dw);
    }

    if (iy1 >= jy1) {
        atomicAdd(dbox1 + 1, -dh);
    } else {
        atomicAdd(dbox2 + 1, -dh);
    }

    if (ix2 <= jx2) {
        atomicAdd(dbox1 + 2, dw);
    } else {
        atomicAdd(dbox2 + 2, dw);
    }

    if (iy2 <= jy2) {
        atomicAdd(dbox1 + 3, dh);
    } else {
        atomicAdd(dbox2 + 3, dh);
    }
}

template <typename T>
__global__ void iou_mn_forward(const int nthreads, const T *boxes1,
                               const T *boxes2, const int m, const int n,
                               T *ious) {
    CUDA_1D_KERNEL_LOOP(index, nthreads) {
        int j = index % n;
        int i = index / n;
        T iou = iou_11(boxes1 + i * 4, boxes2 + j * 4);
        ious[i * m + j] = iou;
    }
}

template <typename T>
__global__ void iou_mn_backward(const int nthreads, T *dboxes1, T *dboxes2,
                                const T *dout, const T *boxes1, const T *boxes2,
                                const int m, const int n, const T *ious) {
    CUDA_1D_KERNEL_LOOP(index, nthreads) {
        int j = index % n;
        int i = index / n;
        iou_11_backward(dboxes1 + i * 4, dboxes2 + j * 4, dout[i * m + j],
                        boxes1 + i * 4, boxes2 + j * 4, ious[i * m + j]);
    }
}

at::Tensor iou_mn_forward_cuda(const at::Tensor &boxes1,
                               const at::Tensor &boxes2) {
    AT_ASSERTM(boxes1.device().is_cuda(), "boxes1 must be a CUDA tensor");
    AT_ASSERTM(boxes2.device().is_cuda(), "boxes2 must be a CUDA tensor");

    // at::TensorArg boxes1_t{boxes1, "boxes1", 1}, boxes2_t{boxes2, "boxes2",
    // 2};

    // at::CheckedFrom c = "iou_mn_forward_cuda";
    // at::checkAllSameGPU(c, {boxes1_t, boxes2_t});
    // at::checkAllSameType(c, {boxes1_t, boxes2_t});

    // at::cuda::CUDAGuard device_guard(boxes1.device());

    auto m = boxes1.size(0);
    auto n = boxes2.size(0);

    at::Tensor ious = at::zeros({m, n}, boxes1.options());

    auto output_size = m * n;
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    dim3 grid(std::min(GET_BLOCKS(512, output_size), 4096));
    dim3 block(512);

    if (ious.numel() == 0) {
        THCudaCheck(hipGetLastError());
        return ious;
    }

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        boxes1.type(), "iou_mn_forward_cuda", [&] {
            iou_mn_forward<scalar_t><<<grid, block, 0, stream>>>(
                output_size, boxes1.contiguous().data<scalar_t>(),
                boxes2.contiguous().data<scalar_t>(), m, n,
                ious.data<scalar_t>());
        });
    THCudaCheck(hipGetLastError());
    return ious;
}

std::tuple<at::Tensor, at::Tensor>
iou_mn_backward_cuda(const at::Tensor &dout, const at::Tensor &boxes1,
                     const at::Tensor &boxes2, const at::Tensor &ious) {
    // Check if input tensors are CUDA tensors
    AT_ASSERTM(dout.device().is_cuda(), "dout must be a CUDA tensor");
    AT_ASSERTM(boxes1.device().is_cuda(), "boxes1 must be a CUDA tensor");
    AT_ASSERTM(boxes2.device().is_cuda(), "boxes2 must be a CUDA tensor");
    AT_ASSERTM(ious.device().is_cuda(), "ious must be a CUDA tensor");

    // at::TensorArg dout_t{dout, "dout", 1}, boxes1_t{boxes1, "boxes1", 2},
    //     boxes2_t{boxes2, "boxes2", 3}, ious_t{ious, "ious", 4};

    // at::CheckedFrom c = "iou_mn_backward_cuda";
    // at::checkAllSameGPU(c, {dout_t, boxes1_t, boxes2_t, ious_t});
    // at::checkAllSameType(c, {dout_t, boxes1_t, boxes2_t, ious_t});

    // at::cuda::CUDAGuard device_guard(dout.device());

    auto m = boxes1.size(0);
    auto n = boxes2.size(0);

    at::Tensor dboxes1 = at::zeros({m, 4}, boxes1.options());
    at::Tensor dboxes2 = at::zeros({n, 4}, boxes2.options());

    auto output_size = m * n;
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    dim3 grid(std::min(GET_BLOCKS(512, output_size), 4096));
    dim3 block(512);

    if (dout.numel() == 0) {
        THCudaCheck(hipGetLastError());
        return std::make_tuple(dboxes1, dboxes2);
    }

    int m_stride = dout.stride(0);
    int n_stride = dout.stride(1);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        dout.type(), "iou_mn_backward_cuda", [&] {
            iou_mn_backward<scalar_t><<<grid, block, 0, stream>>>(
                output_size, dboxes1.data<scalar_t>(), dboxes2.data<scalar_t>(),
                dout.contiguous().data<scalar_t>(),
                boxes1.contiguous().data<scalar_t>(),
                boxes2.contiguous().data<scalar_t>(),
                m, n,
                ious.contiguous().data<scalar_t>());
        });
    THCudaCheck(hipGetLastError());
    return std::make_tuple(dboxes1, dboxes2);
}
